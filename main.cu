#include <cstddef>
#include <iostream>
#include "kernels.cuh"

void initialize(int* data) {
    for (int row=0; row<ROWS; row++) {
        for (int col=0; col<(COLS); col++) {
            data[(row*COLS) + col] = col;
        }
    }
}

void print_matrix(int* matrix) {
    for (int row=0; row<ROWS; row++) {
        printf("%3d [", row);
        for (int col=0; col<(COLS); col++) {
            const auto val = matrix[(row*COLS)+col];
            printf("(%3d) ", val);
        }

        printf("]\n");
    }
}

void format_matrix(int* matrix){

}


int main() {

    // --- Host Memory
    int h_matrix[ROWS * COLS] = {0};
    int h_result[ROWS * COLS] = {0};

    // --- Initialize Data
    initialize(h_matrix);
    // print_matrix(h_matrix);
    std::cout << "\n\n";

    // --- Device Memory
    int* d_matrix;
    int* d_result;
    const size_t matrix_size = sizeof(int) * size_t(ROWS * COLS);

    hipMalloc((void**)&d_matrix, matrix_size);
    hipMalloc((void**)&d_result, matrix_size);
    hipMemcpy(d_matrix, h_matrix, matrix_size, hipMemcpyHostToDevice);

    // --- Kernel Launch
    dim3 block = BLOCK_SIZE;
    dim3 grid {1, 1, 1};

    warp_scan<<<grid, block>>>(d_matrix, d_result);
    hipDeviceSynchronize();

    // --- Copy to Host
    hipMemcpy(h_result, d_result, matrix_size, hipMemcpyDeviceToHost);
    // std::cout << "\n\n";
    // print_matrix(h_result);

    return 0;
}

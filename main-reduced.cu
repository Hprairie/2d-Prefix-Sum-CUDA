#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstddef>
#include <iostream>
#include "kernels.cuh"

void initialize(int* data) {
    for (int row=0; row<ROWS; row++) {
        for (int col=0; col<(COLS); col++) {
            data[(row*COLS) + col] = col;
        }
    }
}

void print_matrix(int* matrix) {
    for (int row=0; row<ROWS; row++) {
        printf("%3d [", row);
        for (int col=0; col<(COLS); col++) {
            const auto val = matrix[(row*COLS)+col];
            printf("(%3d) ", val);
        }

        printf("]\n");
    }
}

void format_matrix(int* matrix){

}


int main() {

    // --- Host Memory
    int h_matrix[ROWS * COLS] = {0};
    int h_result[ROWS * COLS] = {0};

    // --- Initialize Data
    initialize(h_matrix);
    //print_matrix(h_matrix);
    std::cout << "\n\n";

    // --- Device Memory
    int* d_matrix;
    int* d_result;
    const size_t matrix_size = sizeof(int) * size_t(ROWS * COLS);

    hipMalloc((void**)&d_matrix, matrix_size);
    hipMalloc((void**)&d_result, matrix_size);
    hipMemcpy(d_matrix, h_matrix, matrix_size, hipMemcpyHostToDevice);

    //generate smem total
    static constexpr int smem_size = std::max({
            sizeof(typename hipcub::BlockLoad<int, BLOCK_SIZE.x, ITEMS, hipcub::BLOCK_LOAD_DIRECT, BLOCK_SIZE.y>::TempStorage),
            sizeof(typename hipcub::BlockStore<int, BLOCK_SIZE.x, ITEMS, hipcub::BLOCK_STORE_DIRECT, BLOCK_SIZE.y>::TempStorage),
            sizeof(typename hipcub::WarpScan<int, BLOCK_SIZE.x>::TempStorage) * BLOCK_SIZE.y
            });
    std::cout << smem_size << "\n";

    hipStream_t stream = NULL;

    // --- Kernel Launch
    dim3 block = BLOCK_SIZE;
    dim3 grid {1, 1, 1};

    shared_warp_scan<<<grid, block, smem_size, stream>>>(d_matrix, d_result);
    hipDeviceSynchronize();

    // --- Copy to Host
    hipMemcpy(h_result, d_result, matrix_size, hipMemcpyDeviceToHost);
    // std::cout << "\n\n";
    // print_matrix(h_result);

    return 0;
}
